#define EPS 1e-5
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <random>
#include <chrono>
#include <iostream>
//Computes W @ C.T
__global__ void quantize2bitKernel(float *matrix, unsigned int num_elements)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < num_elements)
  {
    __shared__ float gamma_shared;
    if (threadIdx.x == 0)
      gamma_shared = 0.0f;
    __syncthreads();
    gamma_shared += fabs(matrix[idx]);
    __syncthreads();
    if (threadIdx.x == 0)
    {
      gamma_shared /= num_elements;
      gamma_shared += EPS;
    }
    __syncthreads();
    float scaled = matrix[idx] / gamma_shared;
    float abs_scaled = fabs(scaled);
    float rounded = round(abs_scaled);
    rounded = fmin(rounded, 1.0f);
    matrix[idx] = (scaled > 0) ? rounded : -rounded;
  }
}

__device__ __uint8_t convert(const float num)
{
  return (num == -1) ? 0 : (num == 0) ? 1
                                      : 2;
}

__global__ void packingKernel(float *matrix, __uint8_t *res, unsigned int size)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size)
  {
    __uint8_t currRes = 0;
    currRes |= convert(matrix[idx]) << 6;
    currRes |= convert(matrix[idx + 1]) << 4;
    currRes |= convert(matrix[idx + 2]) << 2;
    currRes |= convert(matrix[idx + 3]);
    res[idx / 4] = currRes;
  }
}

__global__ void matmul_2bKernel(float *input, __uint8_t *weight, float *output,
                                size_t input_rows, size_t input_cols, size_t weight_rows, size_t weight_cols)
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < weight_rows * input_cols)
  {
    int i = idx / input_cols;
    int j = idx % input_cols;
    float acc = 0;
    for (int k = 0; k < input_rows; k += 4)
    {
      __uint8_t w = weight[i * weight_cols + k / 4];
      __uint8_t w1 = (w >> 6) & 0x3;
      __uint8_t w2 = (w >> 4) & 0x3;
      __uint8_t w3 = (w >> 2) & 0x3;
      __uint8_t w4 = w & 0x3;

      if (w1 == 0)
      {
        acc -= input[j * input_rows + k];
      }
      else if (w1 == 2)
      {
        acc += input[j * input_rows + k];
      }

      if (w2 == 0)
      {
        acc -= input[j * input_rows + k + 1];
      }
      else if (w2 == 2)
      {
        acc += input[j * input_rows + k + 1];
      }

      if (w3 == 0)
      {
        acc -= input[j * input_rows + k + 2];
      }
      else if (w3 == 2)
      {
        acc += input[j * input_rows + k + 2];
      }

      if (w4 == 0)
      {
        acc -= input[j * input_rows + k + 3];
      }
      else if (w4 == 2)
      {
        acc += input[j * input_rows + k + 3];
      }
    }
    output[i * input_cols + j] = acc;
  }
}

int main(int argc, char *argv[])
{
  int numRuns = 100;
  unsigned int lda = atoi(argv[1]);
  unsigned int n = lda * lda;

  float *h_A, *h_C, *h_output;
  h_A = new float[n];
  h_C = new float[n];
  h_output = new float[n];

  // Randomly generate values for A and C matrices
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> dis(-10.0, 10.0);

  for (unsigned int i = 0; i < n; ++i)
  {
    h_A[i] = dis(gen);
    h_C[i] = dis(gen);
  }

  float *d_A, *d_C, *d_output;
  hipMalloc((void **)&d_A, sizeof(float) * n);
  hipMalloc((void **)&d_C, sizeof(float) * n);
  hipMalloc((void **)&d_output, sizeof(float) * n);

  hipMemcpy(d_A, h_A, sizeof(float) * n, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, sizeof(float) * n, hipMemcpyHostToDevice);
  double totalTime = 0.0;
  for (int i = 0; i < numRuns; i++)
  {
    auto start = std::chrono::steady_clock::now();
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;
    quantize2bitKernel<<<gridSize, blockSize>>>(d_A, n);
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto diff = end - start;
    totalTime += std::chrono::duration<double, std::milli>(diff).count();
  }
  double avgTime = totalTime / numRuns;
  std::cout << "quantize2bitKernel execution time: " << avgTime << " ms" << std::endl;

  // std::cout << "Quantized weight matrix:" << std::endl;
  // float *h_quantized_A;
  // h_quantized_A = new float[n];
  // cudaMemcpy(h_quantized_A, d_A, sizeof(float) * n, cudaMemcpyDeviceToHost);
  // for (unsigned int i = 0; i < n; ++i) {
  //     std::cout << h_quantized_A[i] << " ";
  //     if ((i + 1) % lda == 0) {
  //         std::cout << std::endl;
  //     }
  // }
  // delete[] h_quantized_A;

  // // Print C matrix
  // std::cout << "C matrix:" << std::endl;
  // for (unsigned int i = 0; i < n; ++i) {
  //     std::cout << h_C[i] << " ";
  //     if ((i + 1) % lda == 0) {
  //         std::cout << std::endl;
  //     }
  // }

  __uint8_t *h_packed;
  h_packed = new __uint8_t[n / 4];
  __uint8_t *d_packed;
  hipMalloc((void **)&d_packed, sizeof(__uint8_t) * (n / 4));
  int blockSize = 256;
  totalTime = 0.0;
  for (int i = 0; i < numRuns; i++)
  {
    auto start = std::chrono::steady_clock::now();
    int gridSize = (n + blockSize - 1) / blockSize;
    packingKernel<<<gridSize, blockSize>>>(d_A, d_packed, n);
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto diff = end - start;
    totalTime += std::chrono::duration<double, std::milli>(diff).count();
  }

  avgTime = totalTime / numRuns;
  std::cout << "packingKernel execution time: " << avgTime << " ms" << std::endl;

  int rows = lda;
  int cols = lda;
  totalTime = 0.0;
  blockSize = 256;
  for (int i = 0; i < numRuns; i++)
  {
    auto start = std::chrono::steady_clock::now();
    int gridSize = (rows * cols + blockSize - 1) / blockSize;
    matmul_2bKernel<<<gridSize, blockSize>>>(d_C, d_packed, d_output, rows, cols, rows, cols / 4);
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();
    auto diff = end - start;
    totalTime += std::chrono::duration<double, std::milli>(diff).count();
  }

  avgTime = totalTime / numRuns;
  std::cout << "Average matmul_2bKernel execution time: " << avgTime << " ms" << std::endl;

  // cudaMemcpy(h_output, d_output, sizeof(float) * n, cudaMemcpyDeviceToHost);
  // std::cout << "Output matrix:" << std::endl;
  // cudaMemcpy(h_output, d_output, sizeof(float) * n, cudaMemcpyDeviceToHost);
  // for (unsigned int i = 0; i < n; ++i)
  // {
  //   std::cout << h_output[i] << " ";
  //   if ((i + 1) % lda == 0)
  //   {
  //     std::cout << std::endl;
  //   }
  // }

  delete[] h_A;
  delete[] h_C;
  delete[] h_output;
  delete[] h_packed;
  hipFree(d_A);
  hipFree(d_C);
  hipFree(d_output);
  hipFree(d_packed);

  return 0;
}
